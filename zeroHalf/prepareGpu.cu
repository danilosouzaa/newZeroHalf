#include "hip/hip_runtime.h"
#include "gpulib/gpu.cuh"
//#include "gCut_gpu.cuh"
#include "gSolutionGpu.cuh"


extern "C" {
#include "prepareGpu.h"

}


void setGpuThread(int nGpu)
{
    gpuSetDevice(nGpu);
    int n;
    gpuGetDevice(&n);
    printf("gpu number %d\n", n);
}

int verifyGpu()
{
    int deviceCount = 0;
    //Commands for verify use correct of GPU
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess)
    {
        //printf("hipGetDeviceCount returned %d\n-> %s\n", (int)error_id, hipGetErrorString(error_id));
        //printf("Result = FAIL\n");
        return -1;
        //exit(1);
    }
    if(deviceCount == 0)
    {
        //printf("No GPU found :(");
        exit(1);
        return -1;
    }
    else
    {
        //printf("Found %d GPUs!\n", deviceCount);
        gpuSetDevice(0);
        //printf("GPU 0 initialized!\n");
        return deviceCount;
    }
}

void shuffle_Set(int *vec, int nSetConstrains, int n)
{
    timeval time;
    gettimeofday(&time, NULL);
    srand((time.tv_sec * 1000) + (time.tv_usec / 1000));
    int i, j, aux ;
    int *num_temp = (int*)malloc(sizeof(int)*nSetConstrains);
    int *vec_aux = (int*)malloc(sizeof(int)*nSetConstrains);
    aux  =  n/nSetConstrains;
    for(i = 0; i < aux ; i++)
    {

        for(j = 0 ; j<nSetConstrains; j++)
        {

            num_temp[j] = rand()%RAND_MAX;
            vec_aux[j] = vec[i*nSetConstrains + j];
        }
        bubble_sort(num_temp,vec_aux,nSetConstrains);
        for(j = 0 ; j<nSetConstrains; j++)
        {
            vec[i*nSetConstrains + j] = vec_aux[j];
        }
    }
    free(num_temp);
    free(vec_aux);
}

Cut_gpu* initial_runGPU(Cut_gpu *h_cut, Cut_gpu_aux *cut_aux, int numberMaxConst, int maxDenominator, int precision, int type, int nThreads, int nBlocks)
{
    int deviceCuda;
    hipSetDevice(0);
    deviceCuda = verifyGpu();
    Cut_gpu* out_h_cut;
    int nRuns;
    if(deviceCuda > 0)
    {
        int i, numberC = 0 ;//,j, nCons = h_cut->numberConstrains;

        for(i = 0; i<h_cut->numberConstrains; i++)
        {
            if(h_cut->typeConstraints[i] == RES_RR)
            {
                numberC++;
            }
        }
        //float auxD = ((float)numberC)/((float)nBlocks);
        int nT = numberC;//nCons/10;
        //int nT = ceil(auxD);//nCons/10;
        int nB = 1;
        //int nB = nBlocks;
        nRuns = nT*nB;
//        nRuns = 1000;
//        nB = 10;
//        nT = 100;
        size_t size_solution_r1 =  sizeof(solutionGpu) +
                                   sizeof(TSMult)*(nRuns) +
                                   sizeof(TSConst)*(nRuns) +
                                   sizeof(TSPAux)*(nRuns);

        size_t size_cut = sizeof(Cut_gpu) +
                          sizeof(TCoefficients)*(h_cut->cont) +
                          sizeof(TElements)*(h_cut->cont) +
                          sizeof(TElementsConstraints)*(h_cut->numberConstrains+1) +
                          sizeof(TRightSide)*(h_cut->numberConstrains) +
                          sizeof(TXAsterisc)*(h_cut->numberVariables) +
                          sizeof(TTypeConstraints)*(h_cut->numberConstrains);

        solutionGpu *h_solution_r1 = allocationStructSolution1(h_cut,nRuns); //cpu
        solutionGpu *d_solution_r1 = createGPUsolution1(h_solution_r1, h_cut,nRuns);//gpu
        Cut_gpu *d_cut = createGPUcut(h_cut, h_cut->numberVariables, h_cut->numberConstrains);
        //FASE 1 PODE TIRAR
        hiprandState_t *states;
        hipMalloc((void**)&states, (nRuns)*sizeof(hiprandState_t));
        //FASE 1 PODE TIRA
        unsigned int *h_seed = (unsigned int*)malloc(sizeof(unsigned int)*(nRuns));
        unsigned int *d_seed;
        srand(time(NULL));
        for(i=0; i<(nRuns); i++)
        {
            h_seed[i] = rand()%100000;
        }
        gpuMalloc((void*)&d_seed, sizeof(unsigned int)*(nRuns));
        gpuMemcpy(d_seed, h_seed, sizeof(unsigned int)*(nRuns), hipMemcpyHostToDevice);
        //---------------------------------------------------------------//
        if(type==1)
        {
            runGPUR1<<<nB,nT>>>(d_cut, d_solution_r1, d_seed, states, nT, precision);
        }
        else
        {
            runGPUR1_aleatory<<<nB,nT>>>(d_cut, d_solution_r1, d_seed, states, nT, precision, maxDenominator);
        }
        gpuDeviceSynchronize();

        gpuMemcpy(h_solution_r1, d_solution_r1, size_solution_r1, hipMemcpyDeviceToHost);
        h_solution_r1->SMult = (TSMult*)(h_solution_r1 + 1);
        h_solution_r1->SConst= (TSConst*)(h_solution_r1->SMult + (nRuns));
        h_solution_r1->SPAux = (TSPAux*)(h_solution_r1->SConst + (nRuns));


        gpuMemcpy(h_cut, d_cut, size_cut, hipMemcpyDeviceToHost);
        h_cut->Coefficients = (TCoefficients*)(h_cut + 1);
        h_cut->Elements = (TElements*)(h_cut->Coefficients + (h_cut->cont));
        h_cut->ElementsConstraints = (TElementsConstraints*)(h_cut->Elements + (h_cut->cont));
        h_cut->rightSide = (TRightSide*)(h_cut->ElementsConstraints+ (h_cut->numberConstrains+1));
        h_cut->xAsterisc = (TXAsterisc*)(h_cut->rightSide + (h_cut->numberConstrains));
        h_cut->typeConstraints = (TTypeConstraints*)(h_cut->xAsterisc+ (h_cut->numberVariables));

        gpuFree(d_solution_r1);
        gpuFree(d_cut);
        gpuFree(d_seed);
        gpuFree(states);
        free(h_seed);

        int cont=0;

        //getchar();

        for(i=0; i<nRuns; i++)
        {
            if(h_solution_r1->SConst[i]!=-1)
            {
                //printf("%d %d /%d \n", h_solution_r1->SConst[i], h_solution_r1->SMult[i], h_solution_r1->SPAux[i]);
                cont++;
            }
        }

        if(cont>0)
        {
            printf("Number cuts generated in the phase 1: %d\n", cont);
            out_h_cut = createCutsOfPhaseOne(h_cut, cut_aux, h_solution_r1, cont,precision,nRuns);
            free(h_solution_r1);
            free(h_cut);

        }
        else
        {
            printf("No cuts generate\n");
            free(h_solution_r1);
            //free(h_cut);
            return h_cut;
        }

    }

    return out_h_cut;

}


void returnDimension(int *nB, int *nT, int nRuns)
{

    int blockSize;      // The launch configurator returned block size
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch
    int gridSize;
    int N = nRuns;

    hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize,runGPUR2, 0, N);
    *nB = minGridSize;
    *nT = blockSize;
}


Cut_gpu* second_phase_runGPU(Cut_gpu *h_cut, Cut_gpu_aux *cut_aux, int numberMaxConst, int nRuns, int maxDenominator, int precision, int nB,int nT, int *pos_R1, int szR)
{
    int deviceCuda;
    deviceCuda = verifyGpu();
    int *consR1;
    int *consNR1;
    int *nElemR1;
    hipSetDevice(0);
    Cut_gpu* out_cut_gpu;

    int n_r = 0, n_nr = 0, i;
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        if((h_cut->typeConstraints[i]==RES_RR)||(h_cut->typeConstraints[i]==RES_R1)||(h_cut->typeConstraints[i]==LPC_CGGPU))
        {
            n_r++;
        }
        else
        {
            n_nr++;
        }
    }
    consR1 = (int*)malloc(sizeof(int)*n_r);
    nElemR1 = (int*)malloc(sizeof(int)*n_r);
    consNR1 = (int*)malloc(sizeof(int)*n_nr);

    n_r = 0;
    n_nr = 0;

    for(i=0; i<h_cut->numberConstrains; i++)
    {
        if((h_cut->typeConstraints[i]==RES_RR)||(h_cut->typeConstraints[i]==RES_R1)||(h_cut->typeConstraints[i]==LPC_CGGPU))
        {
            consR1[n_r] = i;
            nElemR1[n_r] = h_cut->ElementsConstraints[i+1] - h_cut->ElementsConstraints[i];
            n_r++;
        }
        else
        {
            if(h_cut->typeConstraints[i]!=LPC_CGGPUR2)
            {
                consNR1[n_nr]=i;
                n_nr++;
            }
        }
    }
    bubble_sort(nElemR1,consR1,n_r);
    int *Similar = returnOrdConstrainsNR(h_cut);
    float *folga = returnFolga(h_cut);
//    printf("%d %d %d\n",nRuns, nT, nB);
//
//    getchar();
    solutionGpu *h_solution_r2 = allocationStructSolution2(h_cut,numberMaxConst,nRuns);
    int *setConstraint = (int*)malloc(sizeof(int)*numberMaxConst*nRuns);
    calcSetConstraint(setConstraint, pos_R1,numberMaxConst, h_cut->numberConstrains, consR1, consNR1, n_r, n_nr, Similar, folga,  nRuns, szR);
    /*int j;
     for(i=0;i<nRuns;i++){
         for(j=0;j<numberMaxConst;j++){
             printf("%d \t", setConstraint[i*numberMaxConst + j]);

         }
         printf("\n");

     }
    */

    shuffle_Set(setConstraint, numberMaxConst, numberMaxConst*nRuns);

    /*  printf("Depois do Shuffle\n");
       for(i=0;i<nRuns;i++){
          printf("%d:", i);
          for(j=0;j<numberMaxConst;j++){
              printf("%d \t", setConstraint[i*numberMaxConst + j]);

          }
          printf("\n");
      }
    */

    if(deviceCuda>0)
    {
        solutionGpu *d_solution;
        Cut_gpu *d_cut;
        int *d_setConstraint;

        int i, j;
//        if(blockSize*minGridSize < nRuns){
//            nRp = nRuns - blockSize*minGridSize;
//        }

        //nB = 10;
        //nT = nRuns/nB;

        // nB = minGridSize;
        // nT = blockSize;

        size_t size_solution =  sizeof(solutionGpu) +
                                sizeof(TSMult)*(nRuns*4) +
                                sizeof(TSConst)*(numberMaxConst*nRuns) +
                                sizeof(TSPAux)*(nRuns);


        size_t size_cut = sizeof(Cut_gpu) +
                          sizeof(TCoefficients)*(h_cut->cont) +
                          sizeof(TElements)*(h_cut->cont) +
                          sizeof(TElementsConstraints)*(h_cut->numberConstrains+1) +
                          sizeof(TRightSide)*(h_cut->numberConstrains) +
                          sizeof(TXAsterisc)*(h_cut->numberVariables) +
                          sizeof(TTypeConstraints)*(h_cut->numberConstrains);

        d_solution = createGPUsolution2(h_solution_r2,h_cut,numberMaxConst,nRuns);
        d_cut = createGPUcut(h_cut,h_cut->numberVariables,h_cut->numberConstrains);

        hiprandState_t *states;
        hipMalloc((void**)&states, (nT*nB)*sizeof(hiprandState_t));

        unsigned int *h_seed = (unsigned int*)malloc(sizeof(unsigned int)*(nT*nB));
        unsigned int *d_seed;
        srand(time(NULL));
        for(i=0; i<(nT*nB); i++)
        {
            h_seed[i] = rand()%100000;
        }
        gpuMalloc((void**)&d_seed, sizeof(unsigned int)*(nT*nB));
        gpuMemcpy(d_seed, h_seed, sizeof(unsigned int)*(nT*nB), hipMemcpyHostToDevice);

        gpuMalloc((void*)&d_setConstraint, sizeof(int)*(numberMaxConst*nRuns));
        gpuMemcpy(d_setConstraint, setConstraint, sizeof(int)*(numberMaxConst*nRuns), hipMemcpyHostToDevice);

        runGPUR2<<<nB,nT>>>(d_cut, d_solution, d_seed, states, numberMaxConst, d_setConstraint, nT,precision,maxDenominator,nRuns);
	//hipSetDevice(0);
        gpuDeviceSynchronize();
        gpuMemcpy(h_solution_r2, d_solution, size_solution, hipMemcpyDeviceToHost);

        h_solution_r2->SMult = (TSMult*)(h_solution_r2+1);
        h_solution_r2->SConst= (TSConst*)(h_solution_r2->SMult + (nRuns*4));
        h_solution_r2->SPAux = (TSPAux*)(h_solution_r2->SConst + (numberMaxConst*nRuns));

        gpuMemcpy(h_cut, d_cut, size_cut, hipMemcpyDeviceToHost);
        h_cut->Coefficients = (TCoefficients*)(h_cut+1);
        h_cut->Elements = (TElements*)(h_cut->Coefficients + h_cut->cont);
        h_cut->ElementsConstraints = (TElementsConstraints*)(h_cut->Elements + h_cut->cont);
        h_cut->rightSide = (TRightSide*)(h_cut->ElementsConstraints + (h_cut->numberConstrains+1));
        h_cut->xAsterisc = (TXAsterisc*)(h_cut->rightSide + (h_cut->numberConstrains));
        h_cut->typeConstraints = (TTypeConstraints*)(h_cut->xAsterisc + (h_cut->numberVariables));

        free(h_seed);
        gpuFree(states);
        gpuFree(d_setConstraint);
        gpuFree(d_cut);
        gpuFree(d_solution);
        gpuFree(d_seed);
        int cont=0;
        //printf("Number constraints: %d\n", h_cut->numberConstrains);
        for(i=0; i<nT; i++)
        {
            for(j=0; j<nB; j++)
            {
                if(h_solution_r2->SConst[0 + i*numberMaxConst + j*numberMaxConst*nT]!=-1)
                {
                    //printf("%d %d %d\n ",h_solution->SSize[i],h_solution->SPos[i],h_solution->SPAux[i]);
                    //printf("u1: %d / %d \t\t u2: %d / %d\n", h_solution->SMult[i], h_solution->SMult[i + 5*h_cut->numberConstrains], h_solution->SMult[i + 10*h_cut->numberConstrains], h_solution->SMult[i + 15*h_cut->numberConstrains]);
                    cont++;
                }
            }
        }
        if(cont>0)
        {
            printf("Number of Cuts in the second phase:%d\n",cont);
            out_cut_gpu = createCutsOfPhaseTwo(h_cut,cut_aux,h_solution_r2,numberMaxConst,cont,precision,nRuns,nT,nB);
            if(out_cut_gpu==NULL)
            {
                free(consR1);
                free(consNR1);
                free(Similar);
                free(folga);
                free(nElemR1);
                free(setConstraint);
                free(h_solution_r2);

                return h_cut;

            }
        }
        else
        {
            free(consR1);
            free(consNR1);
            free(Similar);
            free(folga);
            free(nElemR1);
            free(setConstraint);
            free(h_solution_r2);

            return h_cut;

        }


    }
    free(consR1);
    free(consNR1);
    free(Similar);
    free(folga);
    free(nElemR1);
    free(setConstraint);
    free(h_solution_r2);
    free(h_cut);
    return out_cut_gpu;

}

int contPar(Cut_gpu* h_cut)
{
    int cont = 0,i;
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        if(h_cut->rightSide[i]%2==0)
        {
            cont++;
        }
    }
    return cont;
}

Cut_gpu* phase_zeroHalf(Cut_gpu *h_cut, Cut_gpu_aux *cut_aux,int nConstraintsPerSet, int precision)
{
    //char *matrixNeighborhood;
    int i,j, n_cuts = 0;
//    int szPar = contPar(h_cut);
//    int szImpar = h_cut->numberConstrains -szPar;
    //int *vPar = (int*)malloc(sizeof(int)*szPar);
    //int *vImpar = (int*)malloc(sizeof(int)*(szImpar));
    //matrixNeighborhood = returnMatrixNeighborhood(h_cut);
    //fillParImpar(vPar,vImpar,h_cut);
    listNeigh *zero_list;
    zero_list = returnListNeighborhood(h_cut);
    int nBlocks, nThreads;
    nBlocks = 10;
    nThreads =  50;//szPar/nBlocks;
    int deviceCuda;
    deviceCuda = verifyGpu();
//    for(i=0;i< zero_list->nPos-1;i++){
//            for(j=zero_list->pos[i];j<zero_list->pos[i+1];j++){
//                printf("Restrição %d com %d\n",i,zero_list->list_n[j]);
//            }
//    }
//    printf("Size pos: %d\n", zero_list->nPos);
    int szPerThreads = zero_list->nList/(nBlocks*nThreads) + 1;
    if(deviceCuda>0)
    {
        size_t size_cut = sizeof(Cut_gpu) +
                          sizeof(TCoefficients)*(h_cut->cont) +
                          sizeof(TElements)*(h_cut->cont) +
                          sizeof(TElementsConstraints)*(h_cut->numberConstrains+1) +
                          sizeof(TRightSide)*(h_cut->numberConstrains) +
                          sizeof(TXAsterisc)*(h_cut->numberVariables) +
                          sizeof(TTypeConstraints)*(h_cut->numberConstrains);
        size_t size_list = sizeof(listNeigh) +
                        sizeof(TList)*(zero_list->nList) +
                        sizeof(TPosList)*(zero_list->nPos) ;

        int *h_solution_zero = (int*)malloc(sizeof(int)*nConstraintsPerSet*nBlocks*nThreads);
        int *d_solution_zero ;
        hipMalloc((void**)&d_solution_zero, sizeof(int)*nConstraintsPerSet*nBlocks*nThreads);
        Cut_gpu *d_cut = createGPUcut(h_cut, h_cut->numberVariables, h_cut->numberConstrains);
        listNeigh *d_list = createGPUlist(zero_list);
        runGPU_zeroHalf<<<nBlocks,nThreads>>>(d_cut, d_list, d_solution_zero, szPerThreads,nThreads,precision);
        gpuDeviceSynchronize();


        gpuMemcpy(h_cut, d_cut, size_cut, hipMemcpyDeviceToHost);
        h_cut->Coefficients = (TCoefficients*)(h_cut + 1);
        h_cut->Elements = (TElements*)(h_cut->Coefficients + (h_cut->cont));
        h_cut->ElementsConstraints = (TElementsConstraints*)(h_cut->Elements + (h_cut->cont));
        h_cut->rightSide = (TRightSide*)(h_cut->ElementsConstraints+ (h_cut->numberConstrains+1));
        h_cut->xAsterisc = (TXAsterisc*)(h_cut->rightSide + (h_cut->numberConstrains));
        h_cut->typeConstraints = (TTypeConstraints*)(h_cut->xAsterisc+ (h_cut->numberVariables));
        gpuMemcpy(h_solution_zero, d_solution_zero, sizeof(int)*nConstraintsPerSet*nBlocks*nThreads, hipMemcpyDeviceToHost);

        gpuFree(d_cut);
        gpuFree(d_solution_zero);
        gpuFree(d_list);
        free(zero_list);
        for(i = 0; i< nBlocks*nThreads;i++){
            if(h_solution_zero[i*2]!=-1){
                n_cuts++;
            }
        }

        if(n_cuts>0){
            Cut_gpu *out_h_cut;
            out_h_cut = createCutsOfZeroHalf(h_cut,cut_aux,h_solution_zero,n_cuts,precision,nThreads,nBlocks,nConstraintsPerSet);
            printf("Num cuts zeroHalf: %d\n", n_cuts);
            free(h_solution_zero);
            free(h_cut);
            return (out_h_cut);
                        //incluir o create cuts zeroHalf

        }else{
            printf("NO cuts zeroHalf.\n");
            free(h_solution_zero);
            return (h_cut);

        }




    }

//    free(vImpar);
//    free(vPar);
    //free(matrixNeighborhood);
    return h_cut;

}


Cut_gpu* phase_zeroHalf_2(Cut_gpu *h_cut,Cut_gpu_aux *cut_aux, int nConst, int nRuns, int precision){
    int i,j, n_cuts = 0;
    int nBlocks, nThreads;
    nBlocks = 10;
    nThreads =  500;//szPar/nBlocks;
    int deviceCuda;
    deviceCuda = verifyGpu();
    int szPerThreads = nRuns/(nBlocks*nThreads) + 1;
    if(deviceCuda>0)
    {
        size_t size_cut = sizeof(Cut_gpu) +
                          sizeof(TCoefficients)*(h_cut->cont) +
                          sizeof(TElements)*(h_cut->cont) +
                          sizeof(TElementsConstraints)*(h_cut->numberConstrains+1) +
                          sizeof(TRightSide)*(h_cut->numberConstrains) +
                          sizeof(TXAsterisc)*(h_cut->numberVariables) +
                          sizeof(TTypeConstraints)*(h_cut->numberConstrains);
        int *h_solution_zero = (int*)malloc(sizeof(int)*nConst*nBlocks*nThreads);
        int *d_solution_zero ;
        hipMalloc((void**)&d_solution_zero, sizeof(int)*nConst*nBlocks*nThreads);
        Cut_gpu *d_cut = createGPUcut(h_cut, h_cut->numberVariables, h_cut->numberConstrains);
        hiprandState_t *states;
        hipMalloc((void**)&states, (nThreads*nBlocks)*sizeof(hiprandState_t));

        unsigned int *h_seed = (unsigned int*)malloc(sizeof(unsigned int)*(nThreads*nBlocks));
        unsigned int *d_seed;
        srand(time(NULL));
        for(i=0; i<(nThreads*nBlocks); i++)
        {
            h_seed[i] = rand()%100000;
        }
        gpuMalloc((void**)&d_seed, sizeof(unsigned int)*(nThreads*nBlocks));
        gpuMemcpy(d_seed, h_seed, sizeof(unsigned int)*(nThreads*nBlocks), hipMemcpyHostToDevice);
        runGPU_zeroHalf_2<<<nBlocks,nThreads>>>(d_cut,d_solution_zero, d_seed, states, szPerThreads,nThreads,precision,nConst);
        gpuDeviceSynchronize();

        gpuMemcpy(h_cut, d_cut, size_cut, hipMemcpyDeviceToHost);
        h_cut->Coefficients = (TCoefficients*)(h_cut + 1);
        h_cut->Elements = (TElements*)(h_cut->Coefficients + (h_cut->cont));
        h_cut->ElementsConstraints = (TElementsConstraints*)(h_cut->Elements + (h_cut->cont));
        h_cut->rightSide = (TRightSide*)(h_cut->ElementsConstraints+ (h_cut->numberConstrains+1));
        h_cut->xAsterisc = (TXAsterisc*)(h_cut->rightSide + (h_cut->numberConstrains));
        h_cut->typeConstraints = (TTypeConstraints*)(h_cut->xAsterisc+ (h_cut->numberVariables));
        gpuMemcpy(h_solution_zero, d_solution_zero, sizeof(int)*nConst*nBlocks*nThreads, hipMemcpyDeviceToHost);

        gpuFree(d_cut);
        gpuFree(d_solution_zero);
        gpuFree(d_seed);
        gpuFree(states);
        free(h_seed);
        for(i = 0; i< nBlocks*nThreads;i++){
            if(h_solution_zero[i*nConst]!=-1){
                n_cuts++;
            }
        }
        if(n_cuts>0){
            Cut_gpu *out_h_cut;
            out_h_cut = createCutsOfZeroHalf_2(h_cut,cut_aux,h_solution_zero,n_cuts,precision,nThreads,nBlocks,nConst);
            printf("Num cuts zeroHalf: %d\n", n_cuts);
            free(h_solution_zero);
            free(h_cut);
            return (out_h_cut);
//                        //incluir o create cuts zeroHalf
        }else{
           printf("NO cuts zeroHalf.\n");
           free(h_solution_zero);
           return (h_cut);

        }




    }



}
void fillParImpar(int *vPar,int *vImpar, Cut_gpu *h_cut)
{
    int i, cP=0, cI = 0;
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        if(h_cut->rightSide[i]%2==0)
        {
            vPar[cP] = i;
            cP++;
        }
        else
        {
            vImpar[cI] = i;
            cI++;
        }
    }
}


listNeigh *returnListNeighborhood (Cut_gpu *h_cut)
{
    char *matrixNeighborhood = (char*)malloc(sizeof(char)*h_cut->numberConstrains*h_cut->numberConstrains);
    int *m1 = (int*)malloc(sizeof(int)*h_cut->numberConstrains*h_cut->numberVariables);
    int i,j, k, el, cont_temp = 0;
    memset(m1,0, sizeof(int)*h_cut->numberConstrains*h_cut->numberVariables);
    memset(matrixNeighborhood,0, sizeof(char)*h_cut->numberConstrains*h_cut->numberConstrains);
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        for(j = h_cut->ElementsConstraints[i]; j<h_cut->ElementsConstraints[i+1]; j++)
        {
            el = h_cut->Elements[j];
            m1[el + i*h_cut->numberVariables] = h_cut->Coefficients[j];
        }
    }
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        for(j=0; j<h_cut->numberConstrains; j++)
        {
            for(k = 0; k<h_cut->numberVariables; k++)
            {
                if((i!=j)&&( ((m1[k + i*h_cut->numberVariables]>0)&&(m1[k + j*h_cut->numberVariables]>0)) || ((m1[k + i*h_cut->numberVariables]<0)&&(m1[k + j*h_cut->numberVariables]<0)) ) )
                {
                    matrixNeighborhood[i+j*h_cut->numberConstrains] = 1;
                    if((j>i)&&(h_cut->rightSide[i]%2 != h_cut->rightSide[j]%2)){
                        cont_temp++;
                    }
                    break;
                }
            }
        }
    }

    listNeigh *list_t = AllocationListNeigh(h_cut->numberConstrains,cont_temp);
    //int *novaLista = (int*)malloc(sizeof(int)*cont_temp);
    //int *pos = (int*)malloc(sizeof(int)*h_cut->numberConstrains+1);
    cont_temp = 0;
    list_t->pos[0] = 1;
    for(i=0; i<h_cut->numberConstrains; i++)
    {
        for(j=i+1; j<h_cut->numberConstrains; j++)
        {
            if((matrixNeighborhood[i+j*h_cut->numberConstrains] == 1)&&(h_cut->rightSide[i]%2 != h_cut->rightSide[j]%2))
            {
                list_t->list_n[cont_temp] = j;
                //printf("antes: %d %d \n",i,j);
                cont_temp++;
            }
        }
        list_t->pos[i+1] = cont_temp;
    }

    free(m1);
    free(matrixNeighborhood);
    return list_t;
}


