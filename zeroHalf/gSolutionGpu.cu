#include "hip/hip_runtime.h"
/*
 * gSolution.cu
 *
 *  Created on: 31/03/2017
 *      Author: danilo
 */
#include "gSolutionGpu.cuh"




solutionGpu* createGPUsolution1(solutionGpu* h_solution, Cut_gpu* h_cut, int nRuns)
{

    size_t size_solution =  sizeof(solutionGpu) +
                            sizeof(TSMult)*(nRuns) +
                            sizeof(TSConst)*(nRuns) +
                            sizeof(TSPAux)*(nRuns);

    solutionGpu *d_sol;
    gpuMalloc((void**)&d_sol, size_solution);
    gpuMemset(d_sol,0,size_solution);
    h_solution->SMult = (TSMult*)(d_sol+1);
    h_solution->SConst= (TSConst*)(h_solution->SMult + (nRuns));
    h_solution->SPAux = (TSPAux*)(h_solution->SConst + (nRuns));
    gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
    return d_sol;
}


solutionGpu* createGPUsolution2(solutionGpu* h_solution, Cut_gpu* h_cut,int numberMaxConst, int nRuns)
{

    size_t size_solution =  sizeof(solutionGpu) +
                            sizeof(TSMult)*(nRuns*4) +
                            sizeof(TSConst)*(numberMaxConst*nRuns) +
                            sizeof(TSPAux)*(nRuns);

    solutionGpu *d_sol;
    gpuMalloc((void**)&d_sol, size_solution);
    gpuMemset(d_sol,0,size_solution);
    h_solution->SMult = (TSMult*)(d_sol+1);
    h_solution->SConst= (TSConst*)(h_solution->SMult + (nRuns*4));
    h_solution->SPAux = (TSPAux*)(h_solution->SConst + (numberMaxConst*nRuns));
    gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
    return d_sol;
}


__global__ void runGPUR1(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int nThreads, int precision)
{


    int term = threadIdx.x + blockIdx.x*nThreads;
    __shared__ int *constraints;
    __shared__ int pos;
    hiprand_init(seed[term],term,0,&states[term]);

    int violation = 0,i,j;
    if(threadIdx.x == 0)
    {
        pos = 0;
        constraints = (int*)malloc(sizeof(int)*d_cut->numberConstrains);
        for(i=0; i<d_cut->numberConstrains; i++)
        {
            if(d_cut->typeConstraints[i] == RES_RR)
            {
                constraints[pos] = i;
                pos++;

            }
        }
    }
    __syncthreads();

    int res = constraints[threadIdx.x%pos];
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));

    int n1=-1, d1=-1,el, rhs, aux,value_tes;
    int nBest=-1, dBest=-1, violation_best=0;
    for(j = d_cut->ElementsConstraints[ res ] ; j < d_cut->ElementsConstraints[ res +1 ]; j++)
    {
        d1 = d_cut->Coefficients[j];
        n1 = 1;
        while(n1<d1)
        {
            rhs = 0;
            violation = 0;
            value_tes = 0;
            for(i = d_cut->ElementsConstraints[ res ]; i<d_cut->ElementsConstraints[ res + 1 ]; i++)
            {
                el = d_cut->Elements[i];
                aux = d_cut->Coefficients[i] * n1;
                if( ((aux>0&&d1<0)||(aux<0&&d1>0))&&(aux%d1!=0))
                {
                    aux = (aux/d1) -1;
                }
                else
                {
                    aux = aux/d1;
                }
                //aux = aux< 0 ? (aux/d1) - 1 : aux/d1;
                value_tes += aux*d_cut->xAsterisc[el];
            }
            rhs = d_cut->rightSide[ res ]* n1;
            if( ((rhs>0&&d1<0)||(rhs<0&&d1>0))&&(rhs%d1!=0))
            {
                rhs = (rhs/d1) -1;
            }
            else
            {
                rhs = rhs/d1;
            }

            if(value_tes>rhs*precision)
            {
                violation = value_tes - (rhs*precision);
                if(violation>violation_best)
                {
                    violation_best = violation;
                    nBest=n1;
                    dBest=d1;
                }
            }
            n1++;
        }
    }

    if(violation_best!=0)
    {
        d_solution->SConst[term] = res;
        d_solution->SMult[term] = nBest;
        d_solution->SPAux[term] = dBest;
    }
    else
    {
        d_solution->SConst[term] = -1;
        d_solution->SMult[term] = -1;
        d_solution->SPAux[term] = -1;
    }

    free(Coef);
    if(threadIdx.x == 0)
    {
        free(constraints);
    }
}


__global__ void runGPUR1_aleatory(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int nThreads, int precision,int maxDenominator)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    __shared__ int *constraints;
    __shared__ int pos;
    hiprand_init(seed[term],term,0,&states[term]);

    int violation = 0, cont = 0,i;
    if(threadIdx.x == 0)
    {
        pos = 0;
        constraints = (int*)malloc(sizeof(int)*d_cut->numberConstrains);
        for(i=0; i<d_cut->numberConstrains; i++)
        {
            if(d_cut->typeConstraints[i] == RES_RR)
            {
                constraints[pos] = i;
                pos++;
            }
        }

    }
    __syncthreads();

    int res = constraints[threadIdx.x%pos];
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
    cont = 0;
    int n1=-1, d1=-1,el, rhs, aux,value_tes;
    int nBest=-1, dBest=-1, violation_best=0;
    while((cont<20)&&(violation_best==0))
    {
        cont++;
        d1 = hiprand(&states[term])%maxDenominator + 2;
        n1 = 1;
        while(n1<d1)
        {
            rhs = 0;
            violation = 0;
            value_tes = 0;
            //printf("%d/%d\n",n1,d1);
            for(i = d_cut->ElementsConstraints[ res ]; i<d_cut->ElementsConstraints[ res + 1 ]; i++)
            {
                el = d_cut->Elements[i];
                aux = d_cut->Coefficients[i] * n1;
                if( ((aux>0&&d1<0)||(aux<0&&d1>0))&&(aux%d1!=0))
                {
                    aux = (aux/d1) -1;
                }
                else
                {
                    aux = aux/d1;
                }
                value_tes += aux*d_cut->xAsterisc[el];
            }
            rhs = d_cut->rightSide[ res ]* n1;
            if( ((rhs>0&&d1<0)||(rhs<0&&d1>0))&&(rhs%d1!=0))
            {
                rhs = (rhs/d1) -1;
            }
            else
            {
                rhs = rhs/d1;
            }

            if(value_tes>rhs*precision)
            {
                violation = value_tes - (rhs*precision);
                if(violation>violation_best)
                {
                    violation_best = violation;
                    nBest=n1;
                    dBest=d1;
                }
            }
            n1++;
        }
    }

    if(violation_best!=0)
    {
        d_solution->SConst[term] = res;
        d_solution->SMult[term] = nBest;
        d_solution->SPAux[term] = dBest;
    }
    else
    {
        d_solution->SConst[term] = -1;
        d_solution->SMult[term] = -1;
        d_solution->SPAux[term] = -1;
    }

    free(Coef);
    if(threadIdx.x == 0)
    {
        free(constraints);
    }
}

__global__ void runGPU_zeroHalf(Cut_gpu *d_cut, listNeigh *d_list, int  *d_Solution, int szPerThreads,int nThreads, int precision)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    int i,j, cont = 0, c1, c2,el,rhs = 0, aux, value_tes;
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
    int violation = 0, c1_best = -1,c2_best = -1;
    for(i = term*szPerThreads; i < (term + 1)*szPerThreads; i++)
    {
        value_tes = 0;
        memset(Coef,0,sizeof(int)*d_cut->numberVariables);
        rhs = 0;
        if(i >= d_list->nList )
        {
            break;
        }
        c1 = d_list->list_n[i];
        for(j = 0 ; j < d_list->nPos-1; j++)
        {
            if(i< d_list->pos[j+1])
            {
                c2 = j;
                break;
            }
        }
        __syncthreads();
        for(j = d_cut->ElementsConstraints[ c1 ]; j<d_cut->ElementsConstraints[ c1+ 1]; j++)
        {

            el = d_cut->Elements[j];
            Coef[el] += d_cut->Coefficients[j];
        }
        rhs += d_cut->rightSide[c1];

        for(j = d_cut->ElementsConstraints[ c2 ]; j<d_cut->ElementsConstraints[ c2+ 1]; j++)
        {

            el = d_cut->Elements[j];
            Coef[el] += d_cut->Coefficients[j];
        }
        rhs += d_cut->rightSide[c2];
        for(j=0; j<d_cut->numberVariables; j++)
        {
            aux = Coef[j]<0 ? (Coef[j]/2) - 1 : Coef[j]/2;
            value_tes += aux*d_cut->xAsterisc[j];
        }
        aux = rhs<0 ? rhs/2-1 : rhs/2;
        if((value_tes>aux*precision)&&(value_tes-(aux*precision)>violation))
        {
            violation = value_tes-(aux*precision);
            //printf("violation in gpu: %d\n", violation);
            c1_best = c1;
            c2_best = c2;
        }

        //printf("%d %d\n ", c1,c2);
    }
    __syncthreads();
    d_Solution[term*2] = c1_best;
    d_Solution[term*2+1] = c2_best;
    free(Coef);
    //printf("%d: %d\n",blockIdx.x, szPerThreads);
}


__device__ void shuffle_constraints(int *constraints, int sz, unsigned int *seed, hiprandState_t* states, int term)
{
    hiprand_init(seed[term],term,0,&states[term]);
    //printf("number: %d\n", sz);
    int i, j, t;
    if (sz > 1)
    {

        for (i = sz - 1; i > 0 ; i--)
        {
            j = hiprand(&states[term])%(i+1);
            t = constraints[j];
            constraints[j] = constraints[i];
            constraints[i] = t;
        }
    }
}

__global__ void runGPU_zeroHalf_2(Cut_gpu *d_cut, int *d_Solution, unsigned int *seed, hiprandState_t* states, int szPerThreads, int nThreads, int precision, int nConst)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    int i,j, cont = 0, el, aux, value_tes, ite;
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
    int violation = 0, c1_best = -1,c2_best = -1;
    int *c_best  = (int*)malloc(sizeof(int)*nConst);
    int *c  = (int*)malloc(sizeof(int)*nConst);
    int *constraints = (int*)malloc(sizeof(int)*d_cut->numberConstrains);
    int rhs_t = 0;
    for(i=0;i<nConst;i++){
        c_best[i] = -1;
    }


    for(i=0; i<d_cut->numberConstrains; i++)
    {
        constraints[i] = i;
    }
    for(ite = 0 ; ite < szPerThreads; ite ++)
    {
        rhs_t = 0;
        value_tes = 0;
        shuffle_constraints(constraints,d_cut->numberConstrains,seed,states,term);
        hiprand_init(seed[term],term,0,&states[term]);
        j = hiprand(&states[term])%d_cut->numberConstrains;
        for(i=0; i<nConst-1; i++)
        {
            aux = (j+i)%d_cut->numberConstrains;
            //printf("aux: %d\n",aux);
            c[i] = constraints[aux];
            rhs_t += d_cut->rightSide[ c[i] ];
            //printf("c = %d , rhs: %d\n", c[i],rhs_t);
        }
        if(rhs_t%2==0)
        {
            do
            {
                aux = (aux + 1)%d_cut->numberConstrains;
            }
            while(d_cut->rightSide[constraints[aux]]%2 == 0);

            c[i] = constraints[aux];

        }
        else
        {
            do
            {
                aux = (aux + 1)%d_cut->numberConstrains;
            }
            while(d_cut->rightSide[constraints[aux]]%2 == 1);
            c[i] = constraints[aux];
        }
        rhs_t += d_cut->rightSide[ c[i] ];
        //printf("aux: %d\n",aux);
        memset(Coef,0,sizeof(int)*d_cut->numberVariables);
        for(i=0; i<nConst; i++)
        {
            for(j = d_cut->ElementsConstraints[ c[i] ]; j<d_cut->ElementsConstraints[ c[i] + 1]; j++)
            {
                el = d_cut->Elements[j];
                Coef[el] += d_cut->Coefficients[j];
            }
        }
        for(j=0; j<d_cut->numberVariables; j++)
        {
            aux = Coef[j]<0 ? (Coef[j]/2) - 1 : Coef[j]/2;
            value_tes += aux*d_cut->xAsterisc[j];
        }
        aux = rhs_t<0 ? (rhs_t/2)-1 : rhs_t/2;
        if((value_tes>aux*precision)&&(value_tes-(aux*precision)>violation))
        {
            violation = value_tes-(aux*precision);
            //printf("violation in gpu: %d\n", violation);
            for(i=0; i<nConst; i++)
            {
                c_best[i] = c[i];
            }
        }
    }
    __syncthreads();
    for(i=0; i<nConst; i++)
    {
        d_Solution[term*nConst + i] = c_best[i];
    }

    free(c);
    free(Coef);
    free(c_best);
    free(constraints);
}

__global__ void runGPUR2(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int numberMaxConst, int setConstraint[],int nThreads, int precision, int maxDenominator, int nRuns)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    if(term<nRuns)
    {
        // printf("%d: %d %d %d %d\n",term, setConstraint[term*numberMaxConst + 0],setConstraint[term*numberMaxConst + 1],setConstraint[term*numberMaxConst + 2],setConstraint[term*numberMaxConst + 3]);
        int mult_1, mult_2, rest_a,rest_b, i, j, el, rhs1, rhs2, value_tes, violation = 0, aux, n1_best = -1, n2_best = -1, d1_best = -1, qnt_1, d2_best=-1;//, cont=0;
        hiprand_init(seed[term],term,0,&states[term]);
        int Numerator[20];
        int Denominator[20];
        int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
        int *Coef2 = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
        for(i=0; i<20; i++)
        {
            Denominator[i]= hiprand(&states[term])%maxDenominator + 2;
            Numerator[i] = hiprand(&states[term])%(Denominator[i]-1);
        }
        for(mult_1=0; mult_1<20; mult_1++)
        {
            memset(Coef,0,sizeof(int)*d_cut->numberVariables);
            rhs1 = 0;
            for(rest_a = 0; rest_a< numberMaxConst; rest_a++)
            {
                for(i=d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_a] ]; i<d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_a] + 1]; i++)
                {

                    el = d_cut->Elements[i];
                    Coef[el] += d_cut->Coefficients[i] * Numerator[mult_1];
                }
                rhs1 += d_cut->rightSide[ setConstraint[term*numberMaxConst+rest_a] ] * Numerator[mult_1];
                for(mult_2 = 0; mult_2<20; mult_2++)
                {
                    memset(Coef2,0,sizeof(int)*d_cut->numberVariables);
                    value_tes = 0;
                    rhs2 = 0;
                    for(rest_b = rest_a + 1; rest_b<numberMaxConst; rest_b++)
                    {
                        for(j=d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_b] ]; j<d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_b] + 1]; j++)
                        {
                            el = d_cut->Elements[j];
                            Coef2[el] += d_cut->Coefficients[j] * Numerator[mult_2];
                        }
                        rhs2 += d_cut->rightSide[ setConstraint[term*numberMaxConst + rest_b] ]* Numerator[mult_2];
                    }
                    for(j=0; j<d_cut->numberVariables; j++)
                    {
                        aux = Coef[j]<0 ? Coef[j]/Denominator[mult_1] - 1 : Coef[j]/Denominator[mult_1];
                        value_tes += aux*d_cut->xAsterisc[j];
                        aux = Coef2[j]<0 ? Coef2[j]/Denominator[mult_2] - 1 : Coef2[j]/Denominator[mult_2];
                        value_tes += aux*d_cut->xAsterisc[j];
                    }
                    aux = rhs1<0 ? rhs1/Denominator[mult_1]-1 : rhs1/Denominator[mult_1];
                    aux +=  rhs2<0 ? rhs2/Denominator[mult_2]-1 : rhs2/Denominator[mult_2];


                    if((value_tes>aux*precision)&&(value_tes-(aux*precision)>violation))
                    {
                        violation = value_tes-(aux*precision);
//                        if(violation>precision)
//                        {
//                            printf("AQUIIII!!");
//                            for(i=0; i<numberMaxConst; i++)
//                            {
//                                printf("%d ",setConstraint[term*numberMaxConst + i]);//CPU ja vai ter
//                            }
//                            printf("\n");
//                        }
                        n1_best = Numerator[mult_1];
                        d1_best = Denominator[mult_1];
                        n2_best = Numerator[mult_2];
                        d2_best = Denominator[mult_2];
                        qnt_1 = rest_a;
                    }


                }
            }

        }
        __syncthreads();

        if(violation>0)
        {
            for(i=0; i<numberMaxConst; i++)
            {
                d_solution->SConst[i + threadIdx.x*numberMaxConst + blockIdx.x*numberMaxConst*nThreads] = setConstraint[term*numberMaxConst + i];//CPU ja vai ter
            }

            d_solution->SPAux[threadIdx.x + blockIdx.x*nThreads] = qnt_1;
            d_solution->SMult[0 + threadIdx.x*4 + blockIdx.x*4*nThreads] = n1_best;
            d_solution->SMult[1 + threadIdx.x*4 + blockIdx.x*4*nThreads] = d1_best;
            d_solution->SMult[2 + threadIdx.x*4 + blockIdx.x*4*nThreads] = n2_best;
            d_solution->SMult[3 + threadIdx.x*4 + blockIdx.x*4*nThreads] = d2_best;

        }
        else
        {
            for(i=0; i<numberMaxConst; i++)
            {
                d_solution->SConst[i + threadIdx.x*numberMaxConst + blockIdx.x*numberMaxConst*nThreads] = -1;
            }
            d_solution->SPAux[threadIdx.x + blockIdx.x*nThreads] = 0;
            d_solution->SMult[0 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[1 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[2 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[3 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
        }



        free(Coef);
        free(Coef2);
        __syncthreads();
    }

}
