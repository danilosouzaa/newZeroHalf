#include "hip/hip_runtime.h"
/*
 * gSolution.cu
 *
 *  Created on: 31/03/2017
 *      Author: danilo
 */
#include "gSolutionGpu.cuh"




solutionGpu* createGPUsolution1(solutionGpu* h_solution, Cut_gpu* h_cut, int nRuns)
{

    size_t size_solution =  sizeof(solutionGpu) +
                            sizeof(TSMult)*(nRuns) +
                            sizeof(TSConst)*(nRuns) +
                            sizeof(TSPAux)*(nRuns);

    solutionGpu *d_sol;
    gpuMalloc((void**)&d_sol, size_solution);
    gpuMemset(d_sol,0,size_solution);
    h_solution->SMult = (TSMult*)(d_sol+1);
    h_solution->SConst= (TSConst*)(h_solution->SMult + (nRuns));
    h_solution->SPAux = (TSPAux*)(h_solution->SConst + (nRuns));
    gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
    return d_sol;
}


solutionGpu* createGPUsolution2(solutionGpu* h_solution, Cut_gpu* h_cut,int numberMaxConst, int nRuns)
{

    size_t size_solution =  sizeof(solutionGpu) +
                            sizeof(TSMult)*(nRuns*4) +
                            sizeof(TSConst)*(numberMaxConst*nRuns) +
                            sizeof(TSPAux)*(nRuns);

    solutionGpu *d_sol;
    gpuMalloc((void**)&d_sol, size_solution);
    gpuMemset(d_sol,0,size_solution);
    h_solution->SMult = (TSMult*)(d_sol+1);
    h_solution->SConst= (TSConst*)(h_solution->SMult + (nRuns*4));
    h_solution->SPAux = (TSPAux*)(h_solution->SConst + (numberMaxConst*nRuns));
    gpuMemcpy(d_sol, h_solution, size_solution, hipMemcpyHostToDevice);
    return d_sol;
}


__global__ void runGPUR1(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int nThreads, int precision)
{


    int term = threadIdx.x + blockIdx.x*nThreads;
    __shared__ int *constraints;
    __shared__ int pos;
    hiprand_init(seed[term],term,0,&states[term]);

    int violation = 0,i,j;
    if(threadIdx.x == 0)
    {
        pos = 0;
        constraints = (int*)malloc(sizeof(int)*d_cut->numberConstrains);
        for(i=0; i<d_cut->numberConstrains; i++)
        {
            if(d_cut->typeConstraints[i] == RES_RR)
            {
                constraints[pos] = i;
                pos++;

            }
        }
    }
    __syncthreads();

    int res = constraints[threadIdx.x%pos];
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));

    int n1=-1, d1=-1,el, rhs, aux,value_tes;
    int nBest=-1, dBest=-1, violation_best=0;
    for(j = d_cut->ElementsConstraints[ res ] ; j < d_cut->ElementsConstraints[ res +1 ]; j++)
    {
        d1 = d_cut->Coefficients[j];
        n1 = 1;
        while(n1<d1)
        {
            rhs = 0;
            violation = 0;
            value_tes = 0;
            for(i = d_cut->ElementsConstraints[ res ]; i<d_cut->ElementsConstraints[ res + 1 ]; i++)
            {
                el = d_cut->Elements[i];
                aux = d_cut->Coefficients[i] * n1;
                if( ((aux>0&&d1<0)||(aux<0&&d1>0))&&(aux%d1!=0))
                {
                    aux = (aux/d1) -1;
                }
                else
                {
                    aux = aux/d1;
                }
                //aux = aux< 0 ? (aux/d1) - 1 : aux/d1;
                value_tes += aux*d_cut->xAsterisc[el];
            }
            rhs = d_cut->rightSide[ res ]* n1;
            if( ((rhs>0&&d1<0)||(rhs<0&&d1>0))&&(rhs%d1!=0))
            {
                rhs = (rhs/d1) -1;
            }
            else
            {
                rhs = rhs/d1;
            }

            if(value_tes>rhs*precision)
            {
                violation = value_tes - (rhs*precision);
                if(violation>violation_best)
                {
                    violation_best = violation;
                    nBest=n1;
                    dBest=d1;
                }
            }
            n1++;
        }
    }

    if(violation_best!=0)
    {
        d_solution->SConst[term] = res;
        d_solution->SMult[term] = nBest;
        d_solution->SPAux[term] = dBest;
    }
    else
    {
        d_solution->SConst[term] = -1;
        d_solution->SMult[term] = -1;
        d_solution->SPAux[term] = -1;
    }

    free(Coef);
    if(threadIdx.x == 0)
    {
        free(constraints);
    }
}


__global__ void runGPUR1_aleatory(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int nThreads, int precision,int maxDenominator)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    __shared__ int *constraints;
    __shared__ int pos;
    hiprand_init(seed[term],term,0,&states[term]);

    int violation = 0, cont = 0,i;
    if(threadIdx.x == 0)
    {
        pos = 0;
        constraints = (int*)malloc(sizeof(int)*d_cut->numberConstrains);
        for(i=0; i<d_cut->numberConstrains; i++)
        {
            if(d_cut->typeConstraints[i] == RES_RR)
            {
                constraints[pos] = i;
                pos++;
            }
        }

    }
    __syncthreads();

    int res = constraints[threadIdx.x%pos];
    int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
    cont = 0;
    int n1=-1, d1=-1,el, rhs, aux,value_tes;
    int nBest=-1, dBest=-1, violation_best=0;
    while((cont<20)&&(violation_best==0))
    {
        cont++;
        d1 = hiprand(&states[term])%maxDenominator + 2;
        n1 = 1;
        while(n1<d1)
        {
            rhs = 0;
            violation = 0;
            value_tes = 0;
            //printf("%d/%d\n",n1,d1);
            for(i = d_cut->ElementsConstraints[ res ]; i<d_cut->ElementsConstraints[ res + 1 ]; i++)
            {
                el = d_cut->Elements[i];
                aux = d_cut->Coefficients[i] * n1;
                if( ((aux>0&&d1<0)||(aux<0&&d1>0))&&(aux%d1!=0))
                {
                    aux = (aux/d1) -1;
                }
                else
                {
                    aux = aux/d1;
                }
                value_tes += aux*d_cut->xAsterisc[el];
            }
            rhs = d_cut->rightSide[ res ]* n1;
            if( ((rhs>0&&d1<0)||(rhs<0&&d1>0))&&(rhs%d1!=0))
            {
                rhs = (rhs/d1) -1;
            }
            else
            {
                rhs = rhs/d1;
            }

            if(value_tes>rhs*precision)
            {
                violation = value_tes - (rhs*precision);
                if(violation>violation_best)
                {
                    violation_best = violation;
                    nBest=n1;
                    dBest=d1;
                }
            }
            n1++;
        }
    }

    if(violation_best!=0)
    {
        d_solution->SConst[term] = res;
        d_solution->SMult[term] = nBest;
        d_solution->SPAux[term] = dBest;
    }
    else
    {
        d_solution->SConst[term] = -1;
        d_solution->SMult[term] = -1;
        d_solution->SPAux[term] = -1;
    }

    free(Coef);
    if(threadIdx.x == 0)
    {
        free(constraints);
    }
}

__global__ void runGPU_zeroHalf(Cut_gpu *d_cut, listNeigh *d_list, int  *d_Solution, int szPerThreads,int nThreads){
        int term = threadIdx.x + blockIdx.x*nThreads;

        int i, cont = 0, c1, c2;
        int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
        int violation = 0 , violation_best = 0,c1_best = -1,c2_best = -1;
        for(i = term*szPerThreads; i < (term + 1)*szPerThreads;i++){
            c1 = d_list->list_n[i];
        }
        __syncthreads();
        free(Coef);
        //printf("%d: %d\n",blockIdx.x, szPerThreads);
}

__global__ void runGPUR2(Cut_gpu *d_cut, solutionGpu *d_solution, unsigned int *seed, hiprandState_t* states, int numberMaxConst, int setConstraint[],int nThreads, int precision, int maxDenominator, int nRuns)
{
    int term = threadIdx.x + blockIdx.x*nThreads;
    if(term<nRuns)
    {
       // printf("%d: %d %d %d %d\n",term, setConstraint[term*numberMaxConst + 0],setConstraint[term*numberMaxConst + 1],setConstraint[term*numberMaxConst + 2],setConstraint[term*numberMaxConst + 3]);
        int mult_1, mult_2, rest_a,rest_b, i, j, el, rhs1, rhs2, value_tes, violation = 0, aux, n1_best = -1, n2_best = -1, d1_best = -1, qnt_1, d2_best=-1;//, cont=0;
        hiprand_init(seed[term],term,0,&states[term]);
        int Numerator[20];
        int Denominator[20];
        int *Coef = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
        int *Coef2 = (int*)malloc(sizeof(int)*(d_cut->numberVariables));
        for(i=0; i<20; i++)
        {
            Denominator[i]= hiprand(&states[term])%maxDenominator + 2;
            Numerator[i] = hiprand(&states[term])%(Denominator[i]-1);
        }
        for(mult_1=0; mult_1<20; mult_1++)
        {
            memset(Coef,0,sizeof(int)*d_cut->numberVariables);
            rhs1 = 0;
            for(rest_a = 0; rest_a< numberMaxConst; rest_a++)
            {
                for(i=d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_a] ]; i<d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_a] + 1]; i++)
                {

                    el = d_cut->Elements[i];
                    Coef[el] += d_cut->Coefficients[i] * Numerator[mult_1];
                }
                rhs1 += d_cut->rightSide[ setConstraint[term*numberMaxConst+rest_a] ] * Numerator[mult_1];
                for(mult_2 = 0; mult_2<20; mult_2++)
                {
                    memset(Coef2,0,sizeof(int)*d_cut->numberVariables);
                    value_tes = 0;
                    rhs2 = 0;
                    for(rest_b = rest_a + 1; rest_b<numberMaxConst; rest_b++)
                    {
                        for(j=d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_b] ]; j<d_cut->ElementsConstraints[ setConstraint[term*numberMaxConst + rest_b] + 1]; j++)
                        {
                            el = d_cut->Elements[j];
                            Coef2[el] += d_cut->Coefficients[j] * Numerator[mult_2];
                        }
                        rhs2 += d_cut->rightSide[ setConstraint[term*numberMaxConst + rest_b] ]* Numerator[mult_2];
                    }
                    for(j=0; j<d_cut->numberVariables; j++)
                    {
                        aux = Coef[j]<0 ? Coef[j]/Denominator[mult_1] - 1 : Coef[j]/Denominator[mult_1];
                        value_tes += aux*d_cut->xAsterisc[j];
                        aux = Coef2[j]<0 ? Coef2[j]/Denominator[mult_2] - 1 : Coef2[j]/Denominator[mult_2];
                        value_tes += aux*d_cut->xAsterisc[j];
                    }
                    aux = rhs1<0 ? rhs1/Denominator[mult_1]-1 : rhs1/Denominator[mult_1];
                    aux +=  rhs2<0 ? rhs2/Denominator[mult_2]-1 : rhs2/Denominator[mult_2];


                    if((value_tes>aux*precision)&&(value_tes-(aux*precision)>violation))
                    {
                        violation = value_tes-(aux*precision);
//                        if(violation>precision)
//                        {
//                            printf("AQUIIII!!");
//                            for(i=0; i<numberMaxConst; i++)
//                            {
//                                printf("%d ",setConstraint[term*numberMaxConst + i]);//CPU ja vai ter
//                            }
//                            printf("\n");
//                        }
                        n1_best = Numerator[mult_1];
                        d1_best = Denominator[mult_1];
                        n2_best = Numerator[mult_2];
                        d2_best = Denominator[mult_2];
                        qnt_1 = rest_a;
                    }


                }
            }

        }
        __syncthreads();

        if(violation>0)
        {
            for(i=0; i<numberMaxConst; i++)
            {
                d_solution->SConst[i + threadIdx.x*numberMaxConst + blockIdx.x*numberMaxConst*nThreads] = setConstraint[term*numberMaxConst + i];//CPU ja vai ter
            }

            d_solution->SPAux[threadIdx.x + blockIdx.x*nThreads] = qnt_1;
            d_solution->SMult[0 + threadIdx.x*4 + blockIdx.x*4*nThreads] = n1_best;
            d_solution->SMult[1 + threadIdx.x*4 + blockIdx.x*4*nThreads] = d1_best;
            d_solution->SMult[2 + threadIdx.x*4 + blockIdx.x*4*nThreads] = n2_best;
            d_solution->SMult[3 + threadIdx.x*4 + blockIdx.x*4*nThreads] = d2_best;

        }
        else
        {
            for(i=0; i<numberMaxConst; i++)
            {
                d_solution->SConst[i + threadIdx.x*numberMaxConst + blockIdx.x*numberMaxConst*nThreads] = -1;
            }
            d_solution->SPAux[threadIdx.x + blockIdx.x*nThreads] = 0;
            d_solution->SMult[0 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[1 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[2 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
            d_solution->SMult[3 + threadIdx.x*4 + blockIdx.x*4*nThreads] = -1;
        }



        free(Coef);
        free(Coef2);
        __syncthreads();
    }

}
